#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <random>
#include <cstring>
#include "Hash_Decryptor.cuh"
#include "md5.cuh"
#include "md5-hard.cuh"
#include "md5-medium.cuh"
#include "md5-weak.cuh"
#include "md5-non-parallel.cuh"

using namespace std;

string Hash_Decryptor::generate_hash(string password)
{

    char *hash = new char[32];

    char *device_password;
    char *device_hash;

    int pass_len = password.length();
    char *password_arr = new char[pass_len + 1];

    strcpy(password_arr, password.c_str());
    password_arr[pass_len] = '\0';

    /* Defining memory space for device password and hash */
    hipMalloc((void **)&device_password, (pass_len + 1) * sizeof(char));
    hipMalloc((void **)&device_hash, 32 * sizeof(char));

    hipMemcpy(device_password, password_arr, pass_len * sizeof(char), hipMemcpyHostToDevice);

    hash_gen<<<1, 1>>>(device_password, device_hash);

    hipMemcpy(hash, device_hash, 32 * sizeof(char), hipMemcpyDeviceToHost);

    string str_hash(hash);

    return str_hash;
}

string Hash_Decryptor::non_parallel_func(string hash)
{

    int p_len = 7;

    char *hash_arr = new char[32];
    char *password = new char[p_len];

    bool *match = new bool(false);

    // Converting provided string to character array
    strcpy(hash_arr, hash.c_str());

    char *d_password;
    char *d_hash_arr;
    bool *d_match;

    hipMalloc((void **)&d_password, p_len * sizeof(char));
    hipMalloc((void **)&d_hash_arr, 32 * sizeof(char));
    hipMalloc((void **)&d_match, sizeof(bool));

    hipMemcpy(d_match, match, sizeof(bool), hipMemcpyHostToDevice);
    hipMemcpy(d_hash_arr, hash_arr, sizeof(char) * 32, hipMemcpyHostToDevice);

    non_parallel_non_parallel<<<1, 1>>>(d_hash_arr, d_password);

    // Copying back obtained values into host memory

    hipMemcpy(match, d_match, sizeof(bool), hipMemcpyDeviceToHost);
    hipMemcpy(password, d_password, sizeof(char) * p_len, hipMemcpyDeviceToHost);

    if (match)
    {
        return password;
    }

    else
    {
        string not_found = "Inconclusive attack.";
        return not_found;
    }
}

string Hash_Decryptor::weak_parallel_func(string hash)
{
    int p_len = 7;

    char *hash_arr = new char[32];
    char *password = new char[p_len];

    bool *match = new bool(false);

    /* Calculating total combinations of passwords */
    long long combinations = 0;

    for (int i = 1; i <= p_len; i++)
    {
        combinations += pow(26, i);
    }

    long long blocks = (combinations / 256000);

    // Converting provided string to character array
    strcpy(hash_arr, hash.c_str());

    char *d_password;
    char *d_hash_arr;
    bool *d_match;

    hipMalloc((void **)&d_password, p_len * sizeof(char));
    hipMalloc((void **)&d_hash_arr, 32 * sizeof(char));
    hipMalloc((void **)&d_match, sizeof(bool));

    hipMemcpy(d_match, match, sizeof(bool), hipMemcpyHostToDevice);
    hipMemcpy(d_hash_arr, hash_arr, sizeof(char) * 32, hipMemcpyHostToDevice);

    weak_parallel_weak<<<blocks, 512>>>(d_hash_arr, d_password, d_match);

    // Copying back obtained values into host memory

    hipMemcpy(match, d_match, sizeof(bool), hipMemcpyDeviceToHost);
    hipMemcpy(password, d_password, sizeof(char) * p_len, hipMemcpyDeviceToHost);

    if (match)
    {
        return password;
    }

    else
    {
        string not_found = "Inconclusive attack.";
        return not_found;
    }
}

string Hash_Decryptor::medium_parallel_func(string hash)
{
    int p_len = 7;

    char *hash_arr = new char[32];
    char *password = new char[p_len];

    bool *match = new bool(false);

    /* Calculating total combinations of passwords */
    long long combinations = 0;

    for (int i = 1; i <= p_len; i++)
    {
        combinations += pow(52, i);
    }

    long long blocks = (combinations / 256000);

    // Converting provided string to character array
    strcpy(hash_arr, hash.c_str());

    char *d_password;
    char *d_hash_arr;
    bool *d_match;

    hipMalloc((void **)&d_password, p_len * sizeof(char));
    hipMalloc((void **)&d_hash_arr, 32 * sizeof(char));
    hipMalloc((void **)&d_match, sizeof(bool));

    hipMemcpy(d_match, match, sizeof(bool), hipMemcpyHostToDevice);
    hipMemcpy(d_hash_arr, hash_arr, sizeof(char) * 32, hipMemcpyHostToDevice);

    medium_parallel_medium<<<blocks, 512>>>(d_hash_arr, d_password, d_match);

    // Copying back obtained values into host memory

    hipMemcpy(match, d_match, sizeof(bool), hipMemcpyDeviceToHost);
    hipMemcpy(password, d_password, sizeof(char) * p_len, hipMemcpyDeviceToHost);

    if (match)
    {
        return password;
    }

    else
    {
        string not_found = "Inconclusive attack.";
        return not_found;
    }
}

string Hash_Decryptor::hard_parallel_func(string hash)
{
    int p_len = 7;

    char *hash_arr = new char[32];
    char *password = new char[p_len];

    bool *match = new bool(false);

    /* Calculating total combinations of passwords */
    long long combinations = 0;

    for (int i = 1; i <= p_len; i++)
    {
        combinations += pow(62, i);
    }

    long long blocks = (combinations / 256000);

    // Converting provided string to character array
    strcpy(hash_arr, hash.c_str());

    char *d_password;
    char *d_hash_arr;
    bool *d_match;

    hipMalloc((void **)&d_password, p_len * sizeof(char));
    hipMalloc((void **)&d_hash_arr, 32 * sizeof(char));
    hipMalloc((void **)&d_match, sizeof(bool));

    hipMemcpy(d_match, match, sizeof(bool), hipMemcpyHostToDevice);
    hipMemcpy(d_hash_arr, hash_arr, sizeof(char) * 32, hipMemcpyHostToDevice);

    hard_parallel_hard<<<blocks, 512>>>(d_hash_arr, d_password, d_match);

    // Copying back obtained values into host memory

    hipMemcpy(match, d_match, sizeof(bool), hipMemcpyDeviceToHost);
    hipMemcpy(password, d_password, sizeof(char) * p_len, hipMemcpyDeviceToHost);

    if (match)
    {
        return password;
    }

    else
    {
        string not_found = "Inconclusive attack.";
        return not_found;
    }
}
