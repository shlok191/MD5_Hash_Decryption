#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <random>
#include <cstring>
#include <pybind11/pybind11.h>
#include <codecvt>
#include <pybind11/stl.h>
#include <bits/stdc++.h>
#include "Hash_Decryptor.cuh"
#include "md5.cuh"
#include "md5-hard.cuh"
#include "md5-medium.cuh"
#include "md5-weak.cuh"
#include "md5-non-parallel.cuh"

namespace py = pybind11;

using namespace std;

Hash_Decryptor::Hash_Decryptor()
{
    std::printf("Initialized Hash Decryptor object");
}

const std::string Hash_Decryptor::generate_hash(const std::string &password)
{

    char *hash = new char[32];

    char *device_password;
    char *device_hash;

    int pass_len = password.length();
    char *password_arr = new char[pass_len + 1];

    strcpy(password_arr, password.c_str());
    password_arr[pass_len] = '\0';

    /* Defining memory space for device password and hash */
    hipMalloc((void **)&device_password, (pass_len + 1) * sizeof(char));
    hipMalloc((void **)&device_hash, 32 * sizeof(char));

    hipMemcpy(device_password, password_arr, pass_len * sizeof(char), hipMemcpyHostToDevice);

    hash_gen<<<1, 1>>>(device_password, device_hash);

    hipMemcpy(hash, device_hash, 32 * sizeof(char), hipMemcpyDeviceToHost);

    const std::string str_hash = hash;

    stringstream ss;
    for (size_t i = 0; i < str_hash.length(); i += 2)
    {
        unsigned char byte = std::stoi(str_hash.substr(i, 2), nullptr, 16);
        ss << byte;
    }
    return ss.str();
}

string Hash_Decryptor::non_parallel(const std::string &hash)
{

    int p_len = 7;

    char *hash_arr = new char[32];
    char *password = new char[p_len];

    bool *match = new bool(false);

    // Converting provided string to character array
    strcpy(hash_arr, hash.c_str());

    char *d_password;
    char *d_hash_arr;
    bool *d_match;

    hipMalloc((void **)&d_password, p_len * sizeof(char));
    hipMalloc((void **)&d_hash_arr, 32 * sizeof(char));
    hipMalloc((void **)&d_match, sizeof(bool));

    hipMemcpy(d_match, match, sizeof(bool), hipMemcpyHostToDevice);
    hipMemcpy(d_hash_arr, hash_arr, sizeof(char) * 32, hipMemcpyHostToDevice);

    non_parallel_non_parallel<<<1, 1>>>(d_hash_arr, d_password);

    // Copying back obtained values into host memory

    hipMemcpy(match, d_match, sizeof(bool), hipMemcpyDeviceToHost);
    hipMemcpy(password, d_password, sizeof(char) * p_len, hipMemcpyDeviceToHost);

    if (match)
    {
        return password;
    }

    else
    {
        string not_found = "Inconclusive attack.";
        return not_found;
    }
}

string Hash_Decryptor::weak_parallel(const std::string &hash)
{
    int p_len = 7;

    char *hash_arr = new char[32];
    char *password = new char[p_len];

    bool *match = new bool(false);

    /* Calculating total combinations of passwords */
    long long combinations = 0;

    for (int i = 1; i <= p_len; i++)
    {
        combinations += pow(26, i);
    }

    long long blocks = (combinations / 256000);

    // Converting provided string to character array
    strcpy(hash_arr, hash.c_str());

    char *d_password;
    char *d_hash_arr;
    bool *d_match;

    hipMalloc((void **)&d_password, p_len * sizeof(char));
    hipMalloc((void **)&d_hash_arr, 32 * sizeof(char));
    hipMalloc((void **)&d_match, sizeof(bool));

    hipMemcpy(d_match, match, sizeof(bool), hipMemcpyHostToDevice);
    hipMemcpy(d_hash_arr, hash_arr, sizeof(char) * 32, hipMemcpyHostToDevice);

    weak_parallel_weak<<<blocks, 512>>>(d_hash_arr, d_password, d_match);

    // Copying back obtained values into host memory

    hipMemcpy(match, d_match, sizeof(bool), hipMemcpyDeviceToHost);
    hipMemcpy(password, d_password, sizeof(char) * p_len, hipMemcpyDeviceToHost);

    if (match)
    {
        return password;
    }

    else
    {
        string not_found = "Inconclusive attack.";
        return not_found;
    }
}

string Hash_Decryptor::medium_parallel(const std::string &hash)
{
    int p_len = 7;

    char *hash_arr = new char[32];
    char *password = new char[p_len];

    bool *match = new bool(false);

    /* Calculating total combinations of passwords */
    long long combinations = 0;

    for (int i = 1; i <= p_len; i++)
    {
        combinations += pow(52, i);
    }

    long long blocks = (combinations / 256000);

    // Converting provided string to character array
    strcpy(hash_arr, hash.c_str());

    char *d_password;
    char *d_hash_arr;
    bool *d_match;

    hipMalloc((void **)&d_password, p_len * sizeof(char));
    hipMalloc((void **)&d_hash_arr, 32 * sizeof(char));
    hipMalloc((void **)&d_match, sizeof(bool));

    hipMemcpy(d_match, match, sizeof(bool), hipMemcpyHostToDevice);
    hipMemcpy(d_hash_arr, hash_arr, sizeof(char) * 32, hipMemcpyHostToDevice);

    medium_parallel_medium<<<blocks, 512>>>(d_hash_arr, d_password, d_match);

    // Copying back obtained values into host memory

    hipMemcpy(match, d_match, sizeof(bool), hipMemcpyDeviceToHost);
    hipMemcpy(password, d_password, sizeof(char) * p_len, hipMemcpyDeviceToHost);

    if (match)
    {
        return password;
    }

    else
    {
        string not_found = "Inconclusive attack.";
        return not_found;
    }
}

string Hash_Decryptor::hard_parallel(const std::string &hash)
{
    int p_len = 7;

    char *hash_arr = new char[32];
    char *password = new char[p_len];

    bool *match = new bool(false);

    /* Calculating total combinations of passwords */
    long long combinations = 0;

    for (int i = 1; i <= p_len; i++)
    {
        combinations += pow(62, i);
    }

    long long blocks = (combinations / 256000);

    // Converting provided string to character array
    strcpy(hash_arr, hash.c_str());

    char *d_password;
    char *d_hash_arr;
    bool *d_match;

    hipMalloc((void **)&d_password, p_len * sizeof(char));
    hipMalloc((void **)&d_hash_arr, 32 * sizeof(char));
    hipMalloc((void **)&d_match, sizeof(bool));

    hipMemcpy(d_match, match, sizeof(bool), hipMemcpyHostToDevice);
    hipMemcpy(d_hash_arr, hash_arr, sizeof(char) * 32, hipMemcpyHostToDevice);

    hard_parallel_hard<<<blocks, 512>>>(d_hash_arr, d_password, d_match);

    // Copying back obtained values into host memory

    hipMemcpy(match, d_match, sizeof(bool), hipMemcpyDeviceToHost);
    hipMemcpy(password, d_password, sizeof(char) * p_len, hipMemcpyDeviceToHost);

    if (match)
    {
        return password;
    }

    else
    {
        string not_found = "Inconclusive attack.";
        return not_found;
    }
}

PYBIND11_MODULE(HashClass, m)
{
    py::class_<Hash_Decryptor>(m, "Hash_Decryptor")
        .def(py::init<>())
        .def("generate_hash", &Hash_Decryptor::generate_hash, "Generates an MD5 hash from a given string", py::arg("password") = "Shlok")
        .def("non_parallel", &Hash_Decryptor::non_parallel)
        .def("weak_parallel", &Hash_Decryptor::weak_parallel)
        .def("medium_parallel", &Hash_Decryptor::medium_parallel)
        .def("hard_parallel", &Hash_Decryptor::hard_parallel);
}
