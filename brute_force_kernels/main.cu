#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <random>
#include <string>
#include "md5.cuh"
#include "md5-hard.cuh"

using namespace std;

int main(int argc, char *argv[])
{

    char *password = new char[10];
    char *hash = new char[32];
    char *device_password;
    char *device_hash;
    char *returned_d_password;

    bool *match = new bool(false);
    bool *d_match;

    int p_len = 0;

    /* Accept password for conversion from user */
    if (argc > 1)
    {

        password = argv[1];
    }

    while (password[p_len] != '\0')
    {
        p_len++;
    }

    char *returned_password = new char[p_len];

    /* Calculating total combinations */
    long long combinations = 0;

    for (int i = 1; i <= p_len; i++)
    {

        combinations += pow(62, i);
    }

    long long blocks = (combinations / 256000);

    if (combinations % 256000 != 0)
        blocks++;

    hipMalloc((void **)&device_password, p_len * sizeof(char));
    hipMalloc((void **)&device_hash, 32 * sizeof(char));

    hipMalloc((void **)&returned_d_password, p_len * sizeof(char));
    hipMalloc(&d_match, sizeof(bool));

    hipMemcpy(d_match, match, sizeof(bool), hipMemcpyHostToDevice);
    hipMemcpy(device_password, password, p_len * sizeof(char), hipMemcpyHostToDevice);

    /* Call hash function and output generated hash from given password */
    hash_gen<<<1, 1>>>(device_password, device_hash);
    hipDeviceSynchronize();

    hipMemcpy(hash, device_hash, 32 * sizeof(char), hipMemcpyDeviceToHost);

    hipEvent_t start;
    hipEvent_t stop;

    hipEventCreate(&start);
    hipEventCreate(&stop);

    /* Record start time */
    hipEventRecord(start);

    hard_parallel<<<blocks, 512>>>(device_hash, returned_d_password, d_match);
    hipDeviceSynchronize();
    std::cout << "Final error: " << hipGetLastError() << endl;

    /* Record stop time */
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    hipMemcpy(returned_password, returned_d_password, p_len * sizeof(char), hipMemcpyDeviceToHost);

    /* Get the elapsed time in milliseconds */
    float ms;
    hipEventElapsedTime(&ms, start, stop);

    std::cout << "Total time taken: " << ms << " Original Password: " << returned_password << endl;

    return 0;
}
