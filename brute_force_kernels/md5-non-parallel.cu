#include "hip/hip_runtime.h"
#include <iostream>
#include "md5-non-parallel.cuh"
#include "md5.cuh"

using namespace std;

/*
    Definition:

        Kernel to generate hash from given password

    Parameters:

        1. char *password: password to be converted
        2. char *hash: reference to string pointer to store hash value
*/

__device__ void hash_gen_device_non_parallel(char *password, char *hash)
{

    // -------------------------
    // Defining needed variables
    // -------------------------

    int len = 0;

    union
    {

        unsigned word_chunk[16];
        unsigned char original_chunk[64];
    } block_union;

    while (password[len] != '\0')
        len++;

    /* Defining shift amounts */
    unsigned int s[4][4] = {

        {7, 12, 17, 22}, {5, 9, 14, 20}, {4, 11, 16, 23}, {6, 10, 15, 21}};

    /* Reporting k values */
    unsigned int K[64] =

        {0xd76aa478, 0xe8c7b756, 0x242070db, 0xc1bdceee, 0xf57c0faf, 0x4787c62a,
         0xa8304613, 0xfd469501, 0x698098d8, 0x8b44f7af, 0xffff5bb1, 0x895cd7be,
         0x6b901122, 0xfd987193, 0xa679438e, 0x49b40821, 0xf61e2562, 0xc040b340,
         0x265e5a51, 0xe9b6c7aa, 0xd62f105d, 0x02441453, 0xd8a1e681, 0xe7d3fbc8,
         0x21e1cde6, 0xc33707d6, 0xf4d50d87, 0x455a14ed, 0xa9e3e905, 0xfcefa3f8,
         0x676f02d9, 0x8d2a4c8a, 0xfffa3942, 0x8771f681, 0x6d9d6122, 0xfde5380c,
         0xa4beea44, 0x4bdecfa9, 0xf6bb4b60, 0xbebfbc70, 0x289b7ec6, 0xeaa127fa,
         0xd4ef3085, 0x04881d05, 0xd9d4d039, 0xe6db99e5, 0x1fa27cf8, 0xc4ac5665,
         0xf4292244, 0x432aff97, 0xab9423a7, 0xfc93a039, 0x655b59c3, 0x8f0ccc92,
         0xffeff47d, 0x85845dd1, 0x6fa87e4f, 0xfe2ce6e0, 0xa3014314, 0x4e0811a1,
         0xf7537e82, 0xbd3af235, 0x2ad7d2bb, 0xeb86d391};

    /* Defining addition vectors */
    unsigned int M[4] = {1, 5, 3, 7};
    unsigned int T[4] = {0, 1, 5, 0};

    int b_index = len;

    /* Converting bytes to bits, and then dividing by 512 (chunk size) */
    int t_chunks = (((len + 1) * 64) / 512) + 1;

    /* we need 64 bytes == 512 bits per chunk, giving us total mem need of t_chunks * 64*/
    /* Defining bit array of password with precise memory */

    char *bit_array = new char[t_chunks * 64];

    /* Defining buffers A, B, C, D */
    unsigned int buffers[4] = {0x67452301, 0xEFCDAB89, 0x98BADCFE, 0x10325476};

    // ---------------------------
    // Pre-processing input string
    // ---------------------------

    /* Adding initial password to bit_array */
    for (int i = 0; i < len; i++)
        bit_array[i] = password[i];

    /* Adding indicative char showing end of password and beginning of padding */
    bit_array[len] = (unsigned char)0x80;
    b_index++;

    /* Adding padding of bits to make array length to (length % 512) == 448 */
    int multiple = t_chunks * 64;

    while (b_index < multiple)
    {

        bit_array[b_index] = 0;
        b_index++;
    }

    /* Adding length of original password into remaining bit array */

    unsigned temp_len_holder = (8 * len);
    b_index -= 8;

    MD5_break_down var;
    var.word = temp_len_holder;

    for (int i = 0; i < 4; i++, b_index++)
        bit_array[b_index] = var.chunks[i];

    // ----------------------------------------------------------
    // Processing bit_array (processed password) into hash digest
    // ----------------------------------------------------------

    for (int i = 0; i < t_chunks; i++)
    {

        /* Copying 64*8 = 512 bits of first block into block_union's 64 byte member */

        memcpy(block_union.original_chunk, bit_array + (i * 64), 64);

        unsigned int buff_copy[4];

        buff_copy[0] = buffers[0];
        buff_copy[1] = buffers[1];
        buff_copy[2] = buffers[2];
        buff_copy[3] = buffers[3];

        for (int j = 0; j < 4; j++)
        {

            unsigned int M_addition = M[j];
            unsigned int T_addition = T[j];

            for (int k = 0; k < 16; k++)
            {

                unsigned int F = 0;
                int G = ((M_addition * k) + T_addition) % 16;
                int func_index = ((j * 16) + k);

                if (func_index >= 0 && func_index <= 15)
                {

                    F = (buff_copy[1] & buff_copy[2]) | (~buff_copy[1] & buff_copy[3]);
                }

                else if (func_index >= 16 && func_index <= 31)
                {

                    F = (buff_copy[3] & buff_copy[1]) | (~buff_copy[3] & buff_copy[2]);
                }

                else if (func_index >= 32 && func_index <= 47)
                {

                    F = (buff_copy[1] ^ buff_copy[2] ^ buff_copy[3]);
                }

                else if (func_index >= 48 && func_index <= 63)
                {

                    F = (buff_copy[2] ^ (buff_copy[1] | ~buff_copy[3]));
                }

                F = F + buff_copy[0] + K[func_index] + block_union.word_chunk[G];
                F = (F << s[j][k % 4]) | (F >> (32 - s[j][k % 4]));

                buff_copy[0] = buff_copy[3];
                buff_copy[3] = buff_copy[2];
                buff_copy[2] = buff_copy[1];
                buff_copy[1] = buff_copy[1] + F;
            }
        }

        for (int j = 0; j < 4; j++)
        {

            buffers[j] += buff_copy[j];
        }
    }
    // ----------------------------------------------
    // Converting unsigned integers into hexadecimals
    // ----------------------------------------------

    MD5_break_down final_buffer;
    char const hex_chars[16] = {'0', '1', '2', '3', '4', '5', '6', '7', '8', '9', 'a', 'b', 'c', 'd', 'e', 'f'};

    int hash_count = 0;

    for (int i = 0; i < 4; i++)
    {

        final_buffer.word = buffers[i];

        for (int j = 0; j < 4; j++)
        {

            char const byte = final_buffer.chunks[j];

            hash[hash_count] += hex_chars[(byte & 0xF0) >> 4];
            hash_count++;

            hash[hash_count] += hex_chars[(byte & 0x0F) >> 0];
            hash_count++;
        }
    }

    delete[] bit_array;
}

__global__ void non_parallel_non_parallel(char *hash_digest, char *password)
{

    // -----------------------------
    // Defining all needed variables
    // -----------------------------

    /* Hash will be used to store */
    char *hash = new char[32];

    /* Iteration will store current password iteration */
    char *iteration = new char[1];

    iteration[0] = '`';
    int iter_len = 1;

    // -------------------------------
    // Iterating accross all passwords
    // -------------------------------

    while (1)
    {

        for (int i = 0; i < 32; i++)
            hash[i] = NULL;

        /* We must break out of loop after reaching the final possible password iteration */

        if (iter_len == 7)
        {

            int i;
            for (i = 0; i < 7; i++)
            {

                if (iteration[i] != 'z')
                    break;
            }

            if (i == 7)
            {
                return;
            }
        }

        /* Locating the character to replace */

        int i;

        for (i = (iter_len - 1); i >= 0; i--)
        {

            if (iteration[i] != 'z')
                break;

            iteration[i] = 'a';
        }

        /* All characters equal to z, include new char to begin and set all chars to 'a' */

        if (i == (-1))
        {

            iter_len++;
            char *temp = new char[iter_len + 1];

            for (int i = 0; i < iter_len; i++)
            {
                temp[i] = 'a';
            }

            temp[iter_len] = '\0';

            /* Now, begin points to newly created char array */
            delete[] iteration;
            iteration = temp;
        }

        else
        {

            /* Increment char to next value */
            iteration[i] = (char)((int)iteration[i] + 1);
        }

        // ----------------------------------------------
        // Comparing MD5 hash of iteration to hash_digest
        // ----------------------------------------------

        /* Calculate the hash of iteration */
        hash_gen_device_non_parallel(iteration, hash);

        /* Comparing hash generated to given password's hash */
        for (i = 0; i < 32; i++)
        {
            if (hash[i] != hash_digest[i])
                break;
        }

        /* Found a match! */
        if (i == 32)
        {
            for (int i = 0; i < iter_len; i++)
                password[i] = iteration[i];
            return;
        }
    }
}