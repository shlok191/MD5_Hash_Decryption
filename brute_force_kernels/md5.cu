#include "hip/hip_runtime.h"
// -------------------------------------------------------------
//
// Author: Shlok Sabarwal
// Reference: https://en.wikipedia.org/wiki/MD5
//
// For the implementation of the MD5 function, reference from the
// stated MD5 algorithm on the linked Wikipedia page was taken.
//
// --------------------------------------------------------------

#include "md5.cuh"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <iostream>

using namespace std;

/*
    Definition:

        Kernel to generate hash from given password

    Parameters:

        1. char *password: password to be converted
        2. char *hash: reference to string pointer to store hash value
*/

__global__ void hash_gen(char *password, char *hash)
{

    // -------------------------
    // Defining needed variables
    // -------------------------

    int len = 0;

    while (password[len] != '\0')
        len++;

    union
    {

        unsigned word_chunk[16];
        unsigned char original_chunk[64];
    } block_union;

    /* Defining addition vectors */
    unsigned int M[4] = {1, 5, 3, 7};
    unsigned int T[4] = {0, 1, 5, 0};

    int b_index = len;

    /* Converting bytes to bits, and then dividing by 512 (chunk size) */
    int t_chunks = (((len + 1) * 64) / 512) + 1;

    /* we need 64 bytes == 512 bits per chunk, giving us total mem need of t_chunks * 64*/
    int bit_array_len = t_chunks * 64;

    /* Defining bit array of password with precise memory */
    char *bit_array = new char[bit_array_len];

    /* Defining buffers A, B, C, D */
    unsigned int buffers[4] = {0x67452301, 0xEFCDAB89, 0x98BADCFE, 0x10325476};

    /* Defining shift amounts */
    unsigned int s[4][4] = {

        {7, 12, 17, 22}, {5, 9, 14, 20}, {4, 11, 16, 23}, {6, 10, 15, 21}};

    /* Reporting k values */
    unsigned int K[64] =

        {0xd76aa478, 0xe8c7b756, 0x242070db, 0xc1bdceee, 0xf57c0faf, 0x4787c62a,
         0xa8304613, 0xfd469501, 0x698098d8, 0x8b44f7af, 0xffff5bb1, 0x895cd7be,
         0x6b901122, 0xfd987193, 0xa679438e, 0x49b40821, 0xf61e2562, 0xc040b340,
         0x265e5a51, 0xe9b6c7aa, 0xd62f105d, 0x02441453, 0xd8a1e681, 0xe7d3fbc8,
         0x21e1cde6, 0xc33707d6, 0xf4d50d87, 0x455a14ed, 0xa9e3e905, 0xfcefa3f8,
         0x676f02d9, 0x8d2a4c8a, 0xfffa3942, 0x8771f681, 0x6d9d6122, 0xfde5380c,
         0xa4beea44, 0x4bdecfa9, 0xf6bb4b60, 0xbebfbc70, 0x289b7ec6, 0xeaa127fa,
         0xd4ef3085, 0x04881d05, 0xd9d4d039, 0xe6db99e5, 0x1fa27cf8, 0xc4ac5665,
         0xf4292244, 0x432aff97, 0xab9423a7, 0xfc93a039, 0x655b59c3, 0x8f0ccc92,
         0xffeff47d, 0x85845dd1, 0x6fa87e4f, 0xfe2ce6e0, 0xa3014314, 0x4e0811a1,
         0xf7537e82, 0xbd3af235, 0x2ad7d2bb, 0xeb86d391};

    // ---------------------------
    // Pre-processing input string
    // ---------------------------

    /* Adding initial password to bit_array */

    for (int i = 0; i < len; i++)
        bit_array[i] = password[i];

    /* Adding indicative char showing end of password and beginning of padding */
    bit_array[len] = (unsigned char)0x80;
    b_index++;

    /* Adding padding of bits to make array length to (length % 512) == 448 */
    while ((b_index < ((t_chunks * 512) / 8)))
    {

        bit_array[b_index] = 0;
        b_index++;
    }

    /* Adding length of original password into remaining bit array */

    unsigned temp_len_holder = (8 * len);
    b_index -= 8;

    MD5_break_down var;
    var.word = temp_len_holder;

    for (int i = 0; i < 4; i++, b_index++)
        bit_array[b_index] = var.chunks[i];

    // ----------------------------------------------------------
    // Processing bit_array (processed password) into hash digest
    // ----------------------------------------------------------

    for (int i = 0; i < t_chunks; i++)
    {

        /* Copying 64*8 = 512 bits of first block into block_union's 64 byte member */

        memcpy(block_union.original_chunk, bit_array + (i * 64), 64);

        unsigned int buff_copy[4];

        buff_copy[0] = buffers[0];
        buff_copy[1] = buffers[1];
        buff_copy[2] = buffers[2];
        buff_copy[3] = buffers[3];

        for (int j = 0; j < 4; j++)
        {

            unsigned int M_addition = M[j];
            unsigned int T_addition = T[j];

            for (int k = 0; k < 16; k++)
            {

                unsigned int F = 0;
                int G = ((M_addition * k) + T_addition) % 16;

                if (((j * 16) + k) >= 0 && (((j * 16) + k)) <= 15)
                {

                    F = (buff_copy[1] & buff_copy[2]) | (~buff_copy[1] & buff_copy[3]);
                }

                else if (((j * 16) + k) >= 16 && (((j * 16) + k)) <= 31)
                {

                    F = (buff_copy[3] & buff_copy[1]) | (~buff_copy[3] & buff_copy[2]);
                }

                else if (((j * 16) + k) >= 32 && (((j * 16) + k)) <= 47)
                {

                    F = (buff_copy[1] ^ buff_copy[2] ^ buff_copy[3]);
                }

                else if (((j * 16) + k) >= 48 && (((j * 16) + k)) <= 63)
                {

                    F = (buff_copy[2] ^ (buff_copy[1] | ~buff_copy[3]));
                }

                F = F + buff_copy[0] + K[(j * 16) + k] + block_union.word_chunk[G];
                F = (F << s[j][k % 4]) | (F >> (32 - s[j][k % 4]));

                buff_copy[0] = buff_copy[3];
                buff_copy[3] = buff_copy[2];
                buff_copy[2] = buff_copy[1];
                buff_copy[1] = buff_copy[1] + F;
            }
        }

        for (int j = 0; j < 4; j++)
        {

            buffers[j] += buff_copy[j];
        }
    }

    // ----------------------------------------------
    // Converting unsigned integers into hexadecimals
    // ----------------------------------------------

    MD5_break_down final_buffer;
    char const hex_chars[16] = {'0', '1', '2', '3', '4', '5', '6', '7', '8', '9', 'a', 'b', 'c', 'd', 'e', 'f'};

    int hash_count = 0;

    for (int i = 0; i < 4; i++)
    {

        final_buffer.word = buffers[i];

        for (int j = 0; j < 4; j++)
        {

            char const byte = final_buffer.chunks[j];

            hash[hash_count] += hex_chars[(byte & 0xF0) >> 4];
            hash_count++;

            hash[hash_count] += hex_chars[(byte & 0x0F) >> 0];
            hash_count++;
        }
    }

    delete[] bit_array;
}
