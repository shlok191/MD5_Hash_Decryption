#include "hip/hip_runtime.h"
// -------------------------------------------------------------
//
// Author: Shlok Sabarwal
// 
// CUDA kernel implementation of parallel brute-force 
// attack on given password digest. Used to compare parallel
// vs non-parallel GPU performance 
//
// * Characters considered: [a-z]
//  
// * INFORMATION USED TO CALCULATE BEGINNING AND ENDING INDEX *
//  
// Each thread checks 500 passwords
// Threads / Block: 1024
// Words / Block: 512,000
// Total Blocks: (Total combinations) / 512,000
//
// --------------------------------------------------------------

#include <iostream>
#include "md5-weak.cuh"
#include "md5.cuh"

using namespace std;

/* 
    Definition:  
    
        Kernel to generate hash from given password 
    
    Parameters:

        1. char *password: password to be converted
        2. char *hash: reference to string pointer to store hash value
*/

__device__ void hash_gen_device(char *password, char *hash){

    /* Defining shift amounts */
    unsigned int s[4][4] = { 
            
        {7, 12, 17, 22}, {5,  9, 14, 20},
        {4, 11, 16, 23}, {6, 10, 15, 21}};

    /* Reporting k values */
    unsigned int K[64] = 
            
    {   0xd76aa478, 0xe8c7b756, 0x242070db, 0xc1bdceee, 0xf57c0faf, 0x4787c62a, 
        0xa8304613, 0xfd469501, 0x698098d8, 0x8b44f7af, 0xffff5bb1, 0x895cd7be,
        0x6b901122, 0xfd987193, 0xa679438e, 0x49b40821, 0xf61e2562, 0xc040b340, 
        0x265e5a51, 0xe9b6c7aa, 0xd62f105d, 0x02441453, 0xd8a1e681, 0xe7d3fbc8,
        0x21e1cde6, 0xc33707d6, 0xf4d50d87, 0x455a14ed, 0xa9e3e905, 0xfcefa3f8, 
        0x676f02d9, 0x8d2a4c8a, 0xfffa3942, 0x8771f681, 0x6d9d6122, 0xfde5380c,
        0xa4beea44, 0x4bdecfa9, 0xf6bb4b60, 0xbebfbc70, 0x289b7ec6, 0xeaa127fa, 
        0xd4ef3085, 0x04881d05, 0xd9d4d039, 0xe6db99e5, 0x1fa27cf8, 0xc4ac5665,
        0xf4292244, 0x432aff97, 0xab9423a7, 0xfc93a039, 0x655b59c3, 0x8f0ccc92, 
        0xffeff47d, 0x85845dd1, 0x6fa87e4f, 0xfe2ce6e0, 0xa3014314, 0x4e0811a1,
        0xf7537e82, 0xbd3af235, 0x2ad7d2bb, 0xeb86d391
    };

    /* Defining addition vectors */
    unsigned int M[4] = {1, 5, 3, 7};
    unsigned int T[4] = {0, 1, 5, 0};

    // ------------------------- 
    // Defining needed variables 
    // -------------------------

    union {

        unsigned word_chunk[16];
        unsigned char original_chunk[64];
    }   
    block_union;

    int len = 0;

    while(password[len] != '\0')
        len++;
    

    int b_index = len;
    
    /* Converting bytes to bits, and then dividing by 512 (chunk size) */
    int t_chunks = (((len + 1) * 64) / 512) + 1;

    /* we need 64 bytes == 512 bits per chunk, giving us total mem need of t_chunks * 64*/
    /* Defining bit array of password with precise memory */

    char *bit_array = new char[t_chunks * 64];

    /* Defining buffers A, B, C, D */
    unsigned int buffers[4] = { 0x67452301, 0xEFCDAB89, 0x98BADCFE, 0x10325476 };

    // ---------------------------
    // Pre-processing input string
    // ---------------------------

    /* Adding initial password to bit_array */
    for(int i = 0; i < len; i++) bit_array[i] = password[i];

    /* Adding indicative char showing end of password and beginning of padding */
    bit_array[len] = (unsigned char)0x80;
    b_index++;

    /* Adding padding of bits to make array length to (length % 512) == 448 */
    int multiple = t_chunks*64;

    while(b_index < multiple){

        bit_array[b_index] = 0;
        b_index++;
    }
    
    /* Adding length of original password into remaining bit array */
    
    unsigned temp_len_holder = (8*len);
    b_index -= 8;

    MD5_break_down var;
    var.word = temp_len_holder;
    
    for(int i = 0; i < 4; i++, b_index++) bit_array[b_index] = var.chunks[i];
    
    // ----------------------------------------------------------
    // Processing bit_array (processed password) into hash digest
    // ----------------------------------------------------------

    for(int i = 0; i < t_chunks; i++){
        
        /* Copying 64*8 = 512 bits of first block into block_union's 64 byte member */
        
        memcpy(block_union.original_chunk, bit_array + (i*64), 64);

        unsigned int buff_copy[4];
        
        buff_copy[0] = buffers[0];
        buff_copy[1] = buffers[1];
        buff_copy[2] = buffers[2];
        buff_copy[3] = buffers[3];

        for(int j = 0; j < 4; j++){

            unsigned int M_addition = M[j];
            unsigned int T_addition = T[j]; 
    
            for(int k = 0; k < 16; k++){
                
                unsigned int F = 0;
                int G = ((M_addition*k) + T_addition) % 16;
                int func_index = ((j*16)+k);

                if(func_index >= 0 && func_index <= 15){
                    
                    F = (buff_copy[1] & buff_copy[2]) | (~buff_copy[1] & buff_copy[3]);
                }

                else if(func_index >= 16 && func_index <= 31){

                    F = (buff_copy[3] & buff_copy[1]) | (~buff_copy[3] & buff_copy[2]);
                }

                else if(func_index >= 32 && func_index <= 47){

                    F = (buff_copy[1] ^ buff_copy[2] ^ buff_copy[3]);
                }

                else if(func_index >= 48 && func_index <= 63){

                    F = (buff_copy[2] ^ (buff_copy[1] | ~buff_copy[3]));
                }

                F = F + buff_copy[0] + K[func_index] + block_union.word_chunk[G];
                F = (F << s[j][k%4]) | (F >> (32 - s[j][k%4]));
                
                buff_copy[0] = buff_copy[3];
                buff_copy[3] = buff_copy[2];
                buff_copy[2] = buff_copy[1];
                buff_copy[1] = buff_copy[1] + F;     
            }
        }

        for(int j = 0; j < 4; j++){

            buffers[j] += buff_copy[j];
        }
    }
    // ----------------------------------------------
    // Converting unsigned integers into hexadecimals
    // ----------------------------------------------
    
    MD5_break_down final_buffer;
    char const hex_chars[16] = { '0', '1', '2', '3', '4', '5', '6', '7', '8', '9', 'a', 'b', 'c', 'd', 'e', 'f' };

    int hash_count = 0;

    for(int i = 0; i < 4; i++){

        final_buffer.word = buffers[i];
        
        for(int j = 0; j < 4; j++){
            
            char const byte = final_buffer.chunks[j];

            hash[hash_count] += hex_chars [ ( byte & 0xF0 ) >> 4 ];
            hash_count++;

            hash[hash_count] += hex_chars [ ( byte & 0x0F ) >> 0 ];
            hash_count++;
        }
    }

    delete []bit_array;
}

__device__ int calcWordLength(long long index){
    
    int len = 1;
    
    while(index > pow(26, len)){

        index -= pow(26, len);    
        len++;
    }

    return len;
}

__device__ void calculateWord(char *placeholder, long long index){
    int len = 1;
    
    while(index > pow(26, len)){

        index -= pow(26, len);    
        len++;
    }

    for(int i = 0; i < len; i++) placeholder[i] = 'a';

    for(int i = 0; i < len; i++){

        while(pow(26, len-i-1) < index){

            placeholder[i] = (char)((int)placeholder[i] + 1);
            index -= pow(26, len-1-i);
        }
    }

    placeholder[len] = '\0';
}

__global__ void weak_parallel(char *hash_digest, char *return_val, bool *match){

    // ------------------------- 
    // Defining needed variables 
    // -------------------------

    /* Hash will be used to store */
    char* hash = new char[32];

    /* Calculates index of permutation of password thread begins work on  */
    long long beginIndex = (blockIdx.x*512000) + (threadIdx.x*500);
    
    /* Stores the beginning char array */
    
    int beginLen = calcWordLength(beginIndex);
    char *begin = new char[beginLen+1];
    
    calculateWord(begin, beginIndex);
    
    // -------------------------------
    // Iterating accross all passwords
    // -------------------------------

    int count = 0;

    while(1){
        
        /* Break each thread's execution if match turns true */
        
        if(*match == true){
            
            delete []hash;
            delete []begin;

            return;
        }

        for(int i = 0; i < 32; i++)
            hash[i] = NULL;       
        
        // ----------------------------------------------
        // Comparing MD5 hash of iteration to hash_digest
        // ----------------------------------------------

        /* Calculate the hash of iteration */
        hash_gen_device(begin, hash);
        
        /* Comparing hash generated to given password's hash */
        
        int i;
        for(i = 0; i < 32; i++){
            if(hash[i] != hash_digest[i]) break;
        }

        /* Found a match! */
        if(i == 32){

            for(int i = 0; i < beginLen; i++) return_val[i] = begin[i];
            *match = true;

            delete []hash;
            delete []begin;

            return;
        }

        /* We must break out of loop after reaching the final possible password iteration */
        
        if(count >= 500){

            delete []hash;
            delete []begin;

            return;
        }

        /* Locating the character to replace */
        
        for(i = (beginLen-1); i >= 0; i--){    
        
            if(begin[i] != 'z')     
                break;

            begin[i] = 'a';  
        }

        /* All characters equal to z, include new char to begin and set all chars to 'a' */

        if(i == (-1)){
            
            beginLen++;
            char *temp = new char[beginLen+1];

            for(int i = 0; i < beginLen; i++){
                temp[i] = 'a';
            }

            temp[beginLen] = '\0';
            
            /* Now, begin points to newly created char array */
            delete []begin;
            begin = temp;
        }

        else{

            /* Increment char to next value */
            begin[i] = (char)((int)begin[i] + 1);
        }

        count++;
    }

    delete []hash;
    delete []begin;
}